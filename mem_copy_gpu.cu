
 
#include <hip/hip_runtime.h>
#include <stdio.h>
 #include <stdlib.h>

 // Kernel to give a value
 __global__ void value( int *a ){
   *a = 1;
 }

 // Main program
 int main(void){
   int *a;                     // Host memory
   int *a_dev;                 // Device memory
   int size = sizeof(int);     // size of integer
  
   a = (int *) malloc(size);            // Allocate host memory

   hipMalloc( (void**) &a_dev, size);  // Allocate device memory
      
   value <<<1,1>>> (a_dev);             // Launch kernel on device
   
   // Copy device result back to host
   hipMemcpy( a, a_dev, size, hipMemcpyDeviceToHost );
   
   printf("%d\n",*a);   // Print result
   
   hipFree(a_dev);     // Free device memory

   free(a);             // Free host memory
   
   return 0;
 }


