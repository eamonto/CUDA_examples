
 
#include <hip/hip_runtime.h>
#include <stdio.h>
 #include <stdlib.h>

 #define N 128

 // Kernel to add N integers with N parallel blocks
 __global__ void add_block(int *a, int *b, int *c){
   c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
 }

 // Kernel to add N integers with N parallel threads
 __global__ void add_thread(int *a, int *b, int *c){
   c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
 }


 // Main program
 int main(void){

   int *a,*b,*c;              // Host copies
   int *a_dev,*b_dev,*c_dev;  // Device copies
   int size = N*sizeof(int);
   
   // Allocate host memory
   a = (int *) malloc (size);
   b = (int *) malloc (size);
   c = (int *) malloc (size);
   
   // Allocate device memory
   hipMalloc( (void**)&a_dev, size);
   hipMalloc( (void**)&b_dev, size);
   hipMalloc( (void**)&c_dev, size);
   
   // Initialize
   for (int i=0; i<N; i++){
     a[i] = i;
     b[i] = i;
   }
   
   // Copy inputs to device
   hipMemcpy( a_dev, a, size, hipMemcpyHostToDevice );
   hipMemcpy( b_dev, b, size, hipMemcpyHostToDevice );
   
   { // Parallel sum using threads
     
     // Launch kernel on device
     add_thread <<<1,N>>> (a_dev,b_dev,c_dev);
     
     // Copy device result back to host
     hipMemcpy( c, c_dev, size, hipMemcpyDeviceToHost );
     
     // Print result
     for (int i=0; i<N; i++)
       printf("%d\n",c[i]);
   }
   
   {// Parallel sum using blocks
     
    // Launch kernel on device
     add_block <<<N,1>>> (a_dev,b_dev,c_dev);
     
     // Copy device result back to host
     hipMemcpy( c, c_dev, size, hipMemcpyDeviceToHost );
     
     // Print result
     for (int i=0; i<N; i++)
       printf("%d\n",c[i]);
   }
   
   // Free device memory
   hipFree(a_dev);
   hipFree(b_dev);
   hipFree(c_dev);
   
   // Free host memory
   free(a);
   free(b);
   free(c);
   
   return 0;
 }


