
 
#include <hip/hip_runtime.h>
#include <stdio.h>
 #include <stdlib.h>

 // Kernel to add two integers
 __global__ void add(int *a, int *b, int *c){
   *c = *a + *b;
 }

 // Main program
 int main(void){

   int *a,*b,*c;              // Host copies
   int *a_dev,*b_dev,*c_dev;  // Device copies
   int size = sizeof(int);
   
   // Allocate host memory
   a = (int *) malloc (size);
   b = (int *) malloc (size);
   c = (int *) malloc (size);
   
   // Allocate device memory
   hipMalloc( (void**)&a_dev, size);
   hipMalloc( (void**)&b_dev, size);
   hipMalloc( (void**)&c_dev, size);
   
   // Initialize
   *a = 1;
   *b = 2;
   
   // Copy inputs to device
   hipMemcpy( a_dev, a, size, hipMemcpyHostToDevice );
   hipMemcpy( b_dev, b, size, hipMemcpyHostToDevice );
   
   // Launch kernel on device
   add <<<1,1>>> (a_dev,b_dev,c_dev);
   
   // Copy device result back to host
   hipMemcpy( c, c_dev, size, hipMemcpyDeviceToHost );
   
   // Print result
   printf("%d\n",*c);
   
   // Free device memory
   hipFree(a_dev);
   hipFree(b_dev);
   hipFree(c_dev);
   
   // Free host memory
   free(a);
   free(b);
   free(c);
   
   return 0;
 }
