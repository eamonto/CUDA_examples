
 
#include <hip/hip_runtime.h>
#include <stdio.h>
 #include <stdlib.h>

 #define N 128*256
 #define THREADS_PER_BLOCK 256
 #define N_BLOCKS N/THREADS_PER_BLOCK

 // Kernel to add N integers using threads and blocks
 __global__ void add(int *a, int *b, int *c){
   int index = blockIdx.x * blockDim.x + threadIdx.x;

   c[index] = a[index] + b[index];
 }

 // Main program
 int main(void){
   int *a,*b,*c;              // Host copies
   int *a_dev,*b_dev,*c_dev;  // Device copies
   int size = N*sizeof(int);  // Size of N integer
   
   // Allocate host memory
   a = (int *) malloc (size);
   b = (int *) malloc (size);
   c = (int *) malloc (size);
   
   // Allocate device memory
   hipMalloc( (void**)&a_dev, size);
   hipMalloc( (void**)&b_dev, size);
   hipMalloc( (void**)&c_dev, size);
   
   // Initialize
   for (int i=0; i<N; i++){  
     a[i] = i;
     b[i] = i;
   }
   
   // Copy inputs to device
   hipMemcpy( a_dev, a, size, hipMemcpyHostToDevice );
   hipMemcpy( b_dev, b, size, hipMemcpyHostToDevice );
   
   // Launch kernel on device
   add <<< N_BLOCKS , THREADS_PER_BLOCK >>> (a_dev,b_dev,c_dev);
   
   // Copy device result back to host
   hipMemcpy( c, c_dev, size, hipMemcpyDeviceToHost );
   
   // Print result
   for (int i=0; i<N; i++)
     printf("%d\n",c[i]);
   
   // Free device memory
   hipFree(a_dev);
   hipFree(b_dev);
   hipFree(c_dev);
   
   // Free host memory
   free(a);
   free(b);
   free(c);
   
  return 0;
}

