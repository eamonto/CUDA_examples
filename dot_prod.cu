
 
#include <hip/hip_runtime.h>
#include <stdio.h>
 #include <stdlib.h>

 #define N 256*128
 #define THREADS_PER_BLOCK 128
 #define N_BLOCKS N/THREADS_PER_BLOCK

 // Kernel for dot product
 __global__ void dot( int *a, int *b, int *c ) {

   __shared__ int prod[THREADS_PER_BLOCK]; // Shared memory
   int index = blockIdx.x * blockDim.x + threadIdx.x;
   
   prod[threadIdx.x] = a[index] * b[index];
   
   __syncthreads();  // Threads synchronization
   
   if( threadIdx.x == 0) {
     int par_sum = 0;
     
     for(int i=0; i<THREADS_PER_BLOCK; i++)
       par_sum += prod[threadIdx.x]; // Threads reduction
     
     atomicAdd(c,par_sum); // Blocks reduction
   }
 }


 // Main program
 int main(void){
   int *a,*b,*c;              // Host copies
   int *a_dev,*b_dev,*c_dev;  // Device copies
   int size = N*sizeof(int);  // Size of N integer
   
   // Allocate host memory
   a = (int *) malloc (size);
   b = (int *) malloc (size);
   c = (int *) malloc (sizeof(int));
   
   // Allocate device memory
   hipMalloc( (void**)&a_dev, size);
   hipMalloc( (void**)&b_dev, size);
   hipMalloc( (void**)&c_dev, sizeof(int));
   
   // Initialize
   for (int i=0; i<N; i++){  
     a[i] = 1;
     b[i] = 1;
   }
   *c = 0;
   
   // Copy inputs to device
   hipMemcpy( a_dev, a, size       , hipMemcpyHostToDevice );
   hipMemcpy( b_dev, b, size       , hipMemcpyHostToDevice );
   hipMemcpy( c_dev, c, sizeof(int), hipMemcpyHostToDevice );
   
   // Launch kernel on device
   dot <<< N_BLOCKS , THREADS_PER_BLOCK >>> (a_dev, b_dev, c_dev);
   
   // Copy device result back to host
   hipMemcpy( c, c_dev, sizeof(int), hipMemcpyDeviceToHost );
   
   // Print result
   printf("%d\n",*c);
   
   // Free device memory
   hipFree(a_dev);
   hipFree(b_dev);
   hipFree(c_dev);
   
   // Free host memory
   free(a);
   free(b);
   free(c);
   
   return 0;
 }
